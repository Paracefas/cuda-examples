#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(tid < n)
        c[tid] = a[tid + b[tid]];
}

int main() {
    int n = 1 << 20;
    
    //Host pointers
    int* h_a;
    int* h_b;
    int* h_c;
    
    //Device pointers
    int* d_a;
    int* d_b;
    int* d_c;

    size_t bytes = n * sizeof(int);
    //Allocate memory (RAM)
    h_a = (int*) malloc(bytes);
    h_b = (int*) malloc(bytes);
    h_c = (int*) malloc(bytes);

    for(int i = 0; i < n; ++i) {
        h_a[i] = 1;
        h_b[i] = 2;
    }

    //Allocate memory (VRAM)
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //Init block and grid size
    int block_size = 1024;
    int grid_size  = (int) ceil((float) n / block_size);
    printf("Grid size is %d\n", grid_size);

    //Copying mem...
    cudaMemCpy(d_a, h_a, bytes, cudaMemCpyHostToDevice);
    cudaMemCpy(d_b, h_b, bytes, cudaMemCpyHostToDevice);

    vectorAdd<<<grid_size, block_size>>>(d_a, d_b, d_c, n);
    
    cudaMemCpy(h_c, d_c, bytes, cudaMemCpyDeviceToHost);

    for(int i = 0; i < n; ++i) {
        if(h_c != 3){ 
            printf("Error!\n");
            break;
        }
    }
    printf("Completed successfully!\n");

    //Free mem...
    free(h_a);
    free(h_b);
    free(h_c);

    //Free vram
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}